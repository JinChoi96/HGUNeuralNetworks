/***
	Copyright 2012 Injung Kim

	Licensed under the Apache License, Version 2.0 (the "License");
	you may not use this file except in compliance with the License.
	You may obtain a copy of the License at

		http://www.apache.org/licenses/LICENSE-2.0

	Unless required by applicable law or agreed to in writing, software
	distributed under the License is distributed on an "AS IS" BASIS,
	WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
	See the License for the specific language governing permissions and
	limitations under the License.
***/


#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void Propagate_kernel(float *pDevInput, float *pDevWeight, int inDim, int outDim, float *pDevOutput);

void PropagateOnDevice(float *pInput, float *pWeight, int inDim, int outDim, float *pOutput)
{
	float *pDevInput = NULL, *pDevOutput = NULL, *pDevWeight = NULL;

	// allocate device memory	
	hipMalloc((void**)&pDevInput, inDim * sizeof(float));
	hipMalloc((void**)&pDevOutput, outDim * sizeof(float));
	hipMalloc((void**)&pDevWeight, (inDim+1)*outDim * sizeof(float));
	
	// copy input and weight to device memory
	hipMemcpy(pDevInput, pInput, inDim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pDevWeight, pWeight, (inDim+1)*outDim * sizeof(float), hipMemcpyHostToDevice);
	
	// compute output on CUDA device
	Propagate_kernel<<<1, outDim>>>(pDevInput, pDevWeight, inDim, outDim, pDevOutput);
	
	// copy output to host memory
	hipMemcpy(pOutput, pDevOutput, outDim * sizeof(float), hipMemcpyDeviceToHost);
	
	// deallocate device memory
	hipFree(pDevInput);
	hipFree(pDevOutput);
	hipFree(pDevWeight);
}

__global__ void Propagate_kernel(float *pDevInput, float *pDevWeight, int inDim, int outDim, float *pDevOutput)
{
	// idx is thread index
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	while(idx < outDim){
		float *w = &pDevWeight[idx * (inDim + 1)];

		float net = 0.F;
		for(int i = 0; i < inDim; i++)
				net += pDevInput[i] * w[i];
		net += w[inDim];						// add bias

		pDevOutput[idx] = 1.F/(1.F + (float)exp(-net));

		idx += blockDim.x * gridDim.x; 	//	blockDim.x * gridDim.x is # of threads
	}
}
